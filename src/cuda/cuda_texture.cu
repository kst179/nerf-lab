#include <tiny-cuda-nn/common.h>
#include <loguru/loguru.hpp>

#include "nerf-lab/cuda_texture.h"

namespace nerf {

CudaTexture::CudaTexture(const cv::Mat image) {
    hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(res_desc));

    int bpp;
    hipChannelFormatKind channel_fmt_kind;

    if (image.type() % 8 == CV_32F) {
        bpp = sizeof(float);
        channel_fmt_kind = hipChannelFormatKindFloat;
    } else {
        bpp = sizeof(uint8_t);
        channel_fmt_kind = hipChannelFormatKindUnsigned;
    }

    hipChannelFormatDesc channel_desc;
    channel_desc = hipCreateChannelDesc(bpp, bpp, bpp, bpp, channel_fmt_kind);

    res_desc.resType = hipResourceTypePitch2D;
    res_desc.res.pitch2D.devPtr = image.data;
    res_desc.res.pitch2D.desc = channel_desc;
    res_desc.res.pitch2D.width = image.cols;
    res_desc.res.pitch2D.height = image.rows;
    res_desc.res.pitch2D.pitchInBytes = image.cols * image.channels() * bpp;

    hipTextureDesc tex_desc;
    memset(&tex_desc, 0, sizeof(tex_desc));
    tex_desc.filterMode = hipFilterModeLinear;
    tex_desc.normalizedCoords = false;
    tex_desc.addressMode[0] = hipAddressModeClamp;
    tex_desc.addressMode[1] = hipAddressModeClamp;
    tex_desc.addressMode[2] = hipAddressModeClamp;

    CUDA_CHECK_THROW(hipCreateTextureObject(&texture_, &res_desc, &tex_desc, nullptr));
}

CudaTexture::~CudaTexture() {
    hipError_t error = hipDestroyTextureObject(texture_);
    if (error != hipSuccess) {
        ABORT_F(hipGetErrorString(error));
    }
}

hipTextureObject_t CudaTexture::texture_object() const {
    return texture_;
}

}